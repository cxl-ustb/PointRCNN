#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"
#include "interpolate_gpu.h"

__global__ void three_nn_kernel_fast(int b, int n, int m, float *  unknown, 
    float *  known, float *  dist2, int *  idx) {
    // unknown: (B, N, 3)
    // known: (B, M, 3)
    // output: 
    //      dist2: (B, N, 3)
    //      idx: (B, N, 3)
    
    int bs_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= b || pt_idx >= n) return;

    unknown += bs_idx * n * 3 + pt_idx * 3;
    known += bs_idx * m * 3;
    dist2 += bs_idx * n * 3 + pt_idx * 3;
    idx += bs_idx * n * 3 + pt_idx * 3;

    float ux = unknown[0];
    float uy = unknown[1];
    float uz = unknown[2];

    double best1 = 1e40, best2 = 1e40, best3 = 1e40;
    int besti1 = 0, besti2 = 0, besti3 = 0;
    for (int k = 0; k < m; ++k) {
        float x = known[k * 3 + 0];
        float y = known[k * 3 + 1];
        float z = known[k * 3 + 2];
        float d = (ux - x) * (ux - x) + (uy - y) * (uy - y) + (uz - z) * (uz - z);
        if (d < best1) {
            best3 = best2; besti3 = besti2;
            best2 = best1; besti2 = besti1;
            best1 = d; besti1 = k;
        } 
        else if (d < best2) {
            best3 = best2; besti3 = besti2;
            best2 = d; besti2 = k;
        } 
        else if (d < best3) {
            best3 = d; besti3 = k;
        }
    }
    dist2[0] = best1; dist2[1] = best2; dist2[2] = best3;
    idx[0] = besti1; idx[1] = besti2; idx[2] = besti3;
}

void three_nn_kernel_launcher_fast(int b, int n, int m, float *unknown, 
    float *known, float *dist2, int *idx) {
    // unknown: (B, N, 3)
    // known: (B, M, 3)
    // output: 
    //      dist2: (B, N, 3)
    //      idx: (B, N, 3)

    dim3 blocks(DIVUP(n, THREADS_PER_BLOCK), b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

        PD_DISPATCH_FLOATING_TYPES(
		paddle::DataType::FLOAT32,
        "three_nn_kernel_fast",
        ([&] {
			three_nn_kernel_fast
			<<<blocks, threads, 0>>>
			(b, n, m, unknown, known, dist2, idx);
        }));
}


__global__ void three_interpolate_kernel_fast(int b, int c, int m, int n,float *  points, 
    int *  idx, float *  weight, float *  out) {
    // points: (B, C, M)
    // idx: (B, N, 3)
    // weight: (B, N, 3)
    // output:
    //      out: (B, C, N)

    int bs_idx = blockIdx.z;
    int c_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (bs_idx >= b || c_idx >= c || pt_idx >= n) return;

    weight += bs_idx * n * 3 + pt_idx * 3;
    points += bs_idx * c * m + c_idx * m;
    idx += bs_idx * n * 3 + pt_idx * 3;
    out += bs_idx * c * n + c_idx * n;

    out[pt_idx] = weight[0] * points[idx[0]] + weight[1] * points[idx[1]] + weight[2] * points[idx[2]];
}

void three_interpolate_kernel_launcher_fast(int b, int c, int m, int n, 
    float *points, int *idx, float *weight, float *out) {
    // points: (B, C, M)
    // idx: (B, N, 3)
    // weight: (B, N, 3)
    // output:
    //      out: (B, C, N)

    dim3 blocks(DIVUP(n, THREADS_PER_BLOCK), c, b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);
    PD_DISPATCH_FLOATING_TYPES(
		paddle::DataType::FLOAT32,
        "three_interpolate_kernel_fast",
        ([&] {
			three_interpolate_kernel_fast
			<<<blocks, threads, 0>>>
			(b, c, m, n, points, idx, weight, out);
        }));

}


__global__ void three_interpolate_grad_kernel_fast(int b, int c, int n, int m, float *  grad_out, 
    int *  idx, float *  weight, float *  grad_points) {
    // grad_out: (B, C, N)
    // weight: (B, N, 3)
    // output:
    //      grad_points: (B, C, M)

    int bs_idx = blockIdx.z;
    int c_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (bs_idx >= b || c_idx >= c || pt_idx >= n) return;
    
    grad_out += bs_idx * c * n + c_idx * n + pt_idx;
    weight += bs_idx * n * 3 + pt_idx * 3;
    grad_points += bs_idx * c * m + c_idx * m;
    idx += bs_idx * n * 3 + pt_idx * 3;


    atomicAdd(grad_points + idx[0], grad_out[0] * weight[0]);
    atomicAdd(grad_points + idx[1], grad_out[0] * weight[1]);
    atomicAdd(grad_points + idx[2], grad_out[0] * weight[2]);
}

void three_interpolate_grad_kernel_launcher_fast(
    int b, int c, int n, int m, float *grad_out, 
    int *idx, float *weight, float *grad_points) {
    // grad_out: (B, C, N)
    // weight: (B, N, 3)
    // output:
    //      grad_points: (B, C, M)

    dim3 blocks(DIVUP(n, THREADS_PER_BLOCK), c, b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);
    PD_DISPATCH_FLOATING_TYPES(
		paddle::DataType::FLOAT32,
        "three_interpolate_grad_kernel_fast",
        ([&] {
			three_interpolate_grad_kernel_fast
			<<<blocks, threads, 0>>>
			(b, c, n, m, grad_out, idx, weight, grad_points);
        }));

}